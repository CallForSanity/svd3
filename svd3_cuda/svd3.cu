#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

__global__ void svd3_test()
{
    int tid = blockIdx.x;
    int particleIdx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
    if ( particleIdx >= numParticles ) return;


}
__global__ void svd3_CUDA_test(glm::vec3 v)
{
    int tid = blockIdx.x;

}

void runCudaPart()
{
    // all your cuda code here
    helloCUDA<<<5,1>>>(); // 5 blocks, 1 GPU thread each
    hipDeviceSynchronize();
}
